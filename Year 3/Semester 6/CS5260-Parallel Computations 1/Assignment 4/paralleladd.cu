
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>

__global__ void matrixAddition (int *a, int *b, int *c, int f)
{
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int index = column + row * f;
	if (column < f && row < f)
	{
		c[index] = a[index] + b[index];
	}
}

int main(int argc, char *argv[])
{
	int n = atoi(argv[1]);
	printf("\n\nRunning with n=%d\n", n);
	int a[n][n];
	int b[n][n];
	int c[n][n];
	struct timeval start;
	struct timeval end;
	int *gpu_a;
	int *gpu_b;
	int *gpu_c;
	int matrixSize = n * n * sizeof(int);
	
	//make values
	for(int i = 0; i < n; i++)
	{
		for(int j = 0; j < n; j++)
		{
			a[i][j] = 2 * i + j + 1;
			b[i][j] = i + 4 * j + 2;
		}
	}
	
	hipMalloc((void**)&gpu_a, matrixSize);
	hipMalloc((void**)&gpu_b, matrixSize);
	hipMalloc((void**)&gpu_c, matrixSize);
	
	gettimeofday(&start, NULL);
	hipMemcpy(gpu_a, a, matrixSize, hipMemcpyHostToDevice);
	hipMemcpy(gpu_b, b, matrixSize, hipMemcpyHostToDevice);
	dim3 dimBlock(2,2);
	dim3 dimGrid((int)ceil(n/dimBlock.x),(int)ceil(n/dimBlock.y));
	
	matrixAddition<<<dimGrid,dimBlock>>>(gpu_a,gpu_b,gpu_c,n);
	
	hipMemcpy(c, gpu_c, matrixSize, hipMemcpyDeviceToHost);
	gettimeofday(&end, NULL);
	
	
	hipFree(gpu_a);
	hipFree(gpu_b);
	hipFree(gpu_c);
	
	int success = 1;
	
	for(int i = 0; i < n; i++)
	{
		for(int j = 0; j < n; j++)
		{
			if(c[i][j] != (a[i][j] + b[i][j]))
			{
				success = 0;
				printf("FAILED\n");
				printf("c(%d) != a(%d) + b(%d)\n", c[i][j], a[i][j], b[i][j]);
				break;
			}
		}
		if(success == 0)
		{
			break;
		}
	}
	
	if(success == 1)
	{
		printf("We Did It!!\n");
		int timeran = (((end.tv_sec - start.tv_sec) * 1000000) +(end.tv_usec - start.tv_usec));
		printf("Completed in %d Nano Seconds\n", timeran);
	}
	else
	{
		int timeran = (((end.tv_sec - start.tv_sec) * 1000000) +(end.tv_usec - start.tv_usec));
		printf("Completed in %d Nano Seconds\n", timeran);
	}
	
	return(0);
}
